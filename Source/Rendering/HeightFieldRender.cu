#include "hip/hip_runtime.h"
#pragma once

#include <GL/glew.h>
#include "HeightFieldRender.h"
#include "Framework/Topology/HeightField.h"
#include "Core/Vector.h"
#include "Core/Utility.h"
#include "Framework/Framework/Node.h"
#include "OpenGLContext.h"
#include "Color.h"


namespace PhysIKA
{
	IMPLEMENT_CLASS(HeightFieldRenderModule)

	HeightFieldRenderModule::HeightFieldRenderModule()
		: VisualModule()
		, m_mode(HeightFieldRenderModule::Instance)
		, m_color(Vector3f(0.8, 0.8, 0.8))
	{
	}

	HeightFieldRenderModule::~HeightFieldRenderModule()
	{
	}

	bool HeightFieldRenderModule::initializeImpl()
	{
		Node* parent = getParent();
		if (parent == NULL)
		{
			Log::sendMessage(Log::Error, "Should insert this module into a node!");
			return false;
		}

		auto pSet = TypeInfo::CastPointerDown<HeightField<DataType3f>>(parent->getTopologyModule());
		if (pSet == nullptr)
		{
			Log::sendMessage(Log::Error, "HeightFieldRenderModule: The topology module is not supported!");
			return false;
		}


		Log::sendMessage(Log::Info, "HeightFieldRenderModule successfully initialized!");
	}

	__global__ void PRM_MappingColor(
		DeviceArray<glm::vec3> color,
		DeviceArray<Vector3f> index,
		float minIndex,
		float maxIndex)
	{
		int tId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (tId >= color.size()) return;

		float index_i = index[tId].norm();

		index_i = index_i > maxIndex ? maxIndex : index_i;
		index_i = index_i < minIndex ? minIndex : index_i;

		float a = (index_i - minIndex) / (maxIndex - minIndex);

		Color hsv;
		hsv.HSVtoRGB(240, 1-a, 1);

		color[tId] = glm::vec3(hsv.r, hsv.g, hsv.b);
	}

	__global__ void PRM_MappingColor(
		DeviceArray<glm::vec3> color,
		DeviceArray<float> index,
		float refV,
		float minIndex,
		float maxIndex)
	{
		int tId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (tId >= color.size()) return;

		float index_i = index[tId];

		index_i = index_i > maxIndex ? maxIndex : index_i;
		index_i = index_i < minIndex ? minIndex : index_i;

		float a = (index_i - refV) / (maxIndex - minIndex);

		Color hsv;
		hsv.HSVtoRGB(a * 120 + 120, 1, 1);

		color[tId] = glm::vec3(hsv.r, hsv.g, hsv.b);
	}

	__global__ void SetupTriangles(
		DeviceArray<float3> vertices,
		DeviceArray<float3> normals,
		DeviceArray<float3> colors,
		DeviceArray2D<float> heights,
		float dx,
		float dz,
		float3 origin,
		float3 color)
	{
		int i = threadIdx.x + blockIdx.x * blockDim.x;
		int j = threadIdx.y + blockIdx.y * blockDim.y;

		if (i < heights.Nx() - 1 && j < heights.Ny() - 1)
		{
			int id = i + j * heights.Nx();

			//if (j == 2)
// 			{
// 				printf("%d \n", j);
// 			}

			float3 v1 = origin + make_float3(i*dx, heights(i, j), j*dz);
			float3 v2 = origin + make_float3((i + 1)*dx, heights(i + 1, j), j*dz);
			float3 v3 = origin + make_float3(i*dx, heights(i, j+1), (j+1)*dz);
			float3 v4 = origin + make_float3((i+1)*dx, heights(i+1, j+1), (j+1)*dz);

// 			float3 v1 = origin + make_float3(i*dx, 0.5f, j*dz);
// 			float3 v2 = origin + make_float3((i + 1)*dx, 0.5f, ((j + 1))*dz);
// 			float3 v3 = origin + make_float3(i*dx, 0.5f, (j + 1)*dz);
// 			float3 v4 = origin + make_float3((i + 1)*dx, 0.5f, (j + 1)*dz);

			vertices[3 * (2 * id) + 0] = v1;
			vertices[3 * (2 * id) + 1] = v2;
			vertices[3 * (2 * id) + 2] = v3;

			float3 triN1 = cross(v2 - v1, v3 - v1);
			triN1 = normalize(triN1);

			normals[3 * (2 * id) + 0] = triN1;
			normals[3 * (2 * id) + 1] = triN1;
			normals[3 * (2 * id) + 2] = triN1;

			colors[3 * (2 * id) + 0] = color;
			colors[3 * (2 * id) + 1] = color;
			colors[3 * (2 * id) + 2] = color;


			vertices[3 * (2 * id) + 3] = v3;
			vertices[3 * (2 * id) + 4] = v2;
			vertices[3 * (2 * id) + 5] = v4;

			float3 triN2 = cross(v2 - v3, v4 - v3);
			triN2 = normalize(triN2);

			normals[3 * (2 * id) + 3] = triN2;
			normals[3 * (2 * id) + 4] = triN2;
			normals[3 * (2 * id) + 5] = triN2;

			colors[3 * (2 * id) + 3] = color;
			colors[3 * (2 * id) + 4] = color;
			colors[3 * (2 * id) + 5] = color;
		}

		
	}

	void HeightFieldRenderModule::updateRenderingContext()
	{
		Node* parent = getParent();
		if (parent == NULL)
		{
			Log::sendMessage(Log::Error, "Should insert this module into a node!");
			return;
		}

		auto hf = TypeInfo::CastPointerDown<HeightField<DataType3f>>(parent->getTopologyModule());
		if (hf == nullptr)
		{
			Log::sendMessage(Log::Error, "HeightFieldRenderModule: The topology module is not supported!");
			return;
		}


		auto heights = hf->getHeights();
		int numOfTriangles = (heights.Nx() - 1)*(heights.Ny() - 1) * 2;

		vertices.resize(3 * numOfTriangles);
		normals.resize(3 * numOfTriangles);
		colors.resize(3 * numOfTriangles);

		uint3 total_size;
		total_size.x = heights.Nx() - 1;
		total_size.y = heights.Ny() - 1;
		total_size.z = 1;

		auto ori = hf->getOrigin();

		cuExecute3D(total_size, SetupTriangles,
			vertices,
			normals,
			colors,
			heights,
			hf->getDx(),
			hf->getDz(),
			make_float3(ori[0], ori[1], ori[2]),
			make_float3(1.0, 0.0, 0.0));

		if (m_triangleRender == nullptr)
		{
			m_triangleRender = std::make_shared<TriangleRender>();
		}

		if (m_triangleRender->numberOfTrianlges() != numOfTriangles)
		{
			m_triangleRender->resize(numOfTriangles);
		}
		
		m_triangleRender->setVertexArray(vertices);
		m_triangleRender->setColorArray(colors);
		m_triangleRender->setNormalArray(normals);
	}

	void HeightFieldRenderModule::display()
	{
		glMatrixMode(GL_MODELVIEW_MATRIX);
		glPushMatrix();

		glRotatef(m_rotation.x(), m_rotation.y(), m_rotation.z(), m_rotation.w());
		glTranslatef(m_translation[0], m_translation[1], m_translation[2]);
		glScalef(m_scale[0], m_scale[1], m_scale[2]);

		if (m_triangleRender != nullptr)
			m_triangleRender->display();

		glPopMatrix();
	}

	void HeightFieldRenderModule::setRenderMode(RenderMode mode)
	{
		m_mode = mode;
	}

	void HeightFieldRenderModule::setColor(Vector3f color)
	{
		m_color = color;
	}


	void HeightFieldRenderModule::setReferenceColor(float v)
	{
		m_refV = v;
	}

}